#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <string>
#include <sstream>
#include <vector>
#include <memory>

#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#include <thrust/device_vector.h>
#include <thrust/memory.h>


#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define SIGMOID_0 0
#define LINEAR 1

#define LEARNING_RATE 0.3


using namespace std;

hipblasHandle_t handle;

// ----------------------------------------------------------------------------
//                IO part
// ----------------------------------------------------------------------------


struct IO {
	int inputs, outputs, samples;
	const char *infile,*outfile;

    float *input;
    float *output;

		void read_i(const char* infile) {
			ifstream ifile(infile);

			for(int i=0;i<samples;i++) {
				string line;
				getline(ifile,line);
				istringstream is(line);
				char c;
				for(int p=0;p<inputs;p++) {
					is>>input[IDX2C(p,i,inputs)]>>c;
				}
			}
		}

		void read_o(const char *outfile) {
			ifstream ofile(outfile);
				
			for(int i=0;i<samples;i++) {
				string line;
				getline(ofile,line);
				istringstream is(line);
				char c;
				for(int p=0;p<outputs;p++) {
					is>>output[IDX2C(p,i,outputs)]>>c;
				}
			}
		}

		void read_io(const char* infile, const char* outfile) {
			read_i(infile);
			read_o(outfile);
		}


	IO(int _inputs, int _outputs, int _samples, const char* _infile, const char* _outfile) : inputs(_inputs), outputs(_outputs), samples(_samples),
		infile(_infile), outfile(_outfile) {
		input=new float[inputs*samples];
		output=new float[outputs*samples];
		read_io(infile,outfile);
	}
};

class Autoencoder;
class Layer;
class Matrix;
class Vector;
typedef Layer* layer_ptr;
typedef Matrix* matrix_ptr;
typedef Vector* vector_ptr;
typedef Autoencoder* autoencoder_ptr;

class Matrix {
public:
	int rows;
	int cols;
	thrust::device_vector<float> d_data;
	Matrix(int _rows, int _cols) : rows(_rows), cols(_cols), d_data(rows*cols) {
		thrust::fill(d_data.begin(), d_data.end(), 0.f);
/*
		for(int i=0;i<numel();i++) {
			d_data[i]=0.f;
		}
*/
	}
	Matrix(int _size) : rows(_size), cols(_size), d_data(_size*_size) {
		thrust::fill(d_data.begin(),d_data.end(), 0.f);
/*
		for(int i=0;i<numel();i++) {
			d_data[i]=0.f;
		}
*/
		for(int i=0;i<rows;i++)
			d_data[IDX2C(i,i,rows)]=1.f;
	}
	__host__ __device__ int numel() const {return rows*cols;}
	virtual ~Matrix() {
		d_data.clear();
		d_data.shrink_to_fit();
	}

	void print() {
		thrust::host_vector<float> hv(d_data.begin(), d_data.end());
		for(int i=0;i<rows;i++) {
			for(int j=0;j<cols;j++) {
				printf("%f,",hv[IDX2C(i,j,rows)]);
			}
//			printf("\n");
		}
	}
	void printd() {
		thrust::host_vector<float> hv(d_data.begin(), d_data.end());
		for(int i=0;i<rows;i++) {
			for(int j=0;j<cols;j++) {
				printf("%.0f,",hv[IDX2C(i,j,rows)]);
			}
//			printf("\n");
		}
	}

	void randomize() {
		for(int i=0;i<numel();i++) {
			d_data[i]=(((float)rand())/RAND_MAX)/5.;
		}
	}

	
};


class Vector : public Matrix {
public:
	Vector(int rows, int cols): Matrix(rows, cols) {
	}	
	virtual ~Vector() {
	}
	virtual float operator[](const int idx) = 0;
	virtual void set(int idx,float a) = 0;
	void set(Vector &v) {
		for(int i=0;i<numel() && i<v.numel(); i++) {
			set(i, v[i]);
		}
	}
};

class ColVector : public Vector {
public:
	ColVector(int _size) : Vector(_size, 1) {
	}
	virtual ~ColVector() {
	}

	virtual float operator[](const int idx) {
			return d_data[IDX2C(idx, 0, rows)];
	}
	virtual void set(int idx, float a) {
		d_data[IDX2C(idx,0,rows)]=a;
	}
};

class RowVector : public Vector {
public:
	RowVector(int _size) : Vector(1,_size) {
	}
	virtual ~RowVector() {
	}
	virtual float operator[](const int idx) {
			return d_data[IDX2C(0,idx, rows)];
	}
	virtual void set(int idx, float a) {
		d_data[IDX2C(0,idx,rows)]=a;
	}
};

__device__ inline float sigmoid(float signal) {
	return 1./(1+exp(-1.*signal));
}

__device__ inline float sigmoid_derived(float signal) {
	float s=sigmoid(signal);
	return s*(1-s);
}

__global__ void cuSigmoid(int numel,float* v, float* output) {
	int vIdx=threadIdx.x;

	while(vIdx<numel) {
		output[vIdx]=sigmoid(v[vIdx]);
		vIdx+=blockDim.x;
	}
}

__global__ void cuSigmoidDerived(int numel, float* v, float* outputDerived,int rows) {
	int vIdx=threadIdx.x;

	while(vIdx<numel) {
		outputDerived[IDX2C(vIdx,vIdx, rows)]=sigmoid_derived(v[vIdx]);
		vIdx+=blockDim.x;
	}
}

void sigmoid(vector_ptr v, vector_ptr output, matrix_ptr outputDerived, bool last) {
/*
	for(int i=0;i<v->numel();i++) {
		outputDerived->d_data[IDX2C(i,i, outputDerived->rows)]=sigmoid_derived(v->d_data[i]);
		output->d_data[i]=sigmoid(v->d_data[i]);
	}
*/
	cuSigmoid<<<1,16>>>(v->numel()-(last?0:1), thrust::raw_pointer_cast(v->d_data.data()), thrust::raw_pointer_cast(output->d_data.data()));
	cuSigmoidDerived<<<1,16>>>(v->numel()-(last?0:1),thrust::raw_pointer_cast(v->d_data.data()), thrust::raw_pointer_cast(outputDerived->d_data.data()), outputDerived->rows);
	if(!last) {
		output->d_data[v->numel()-1]=1.f;
		outputDerived->d_data[IDX2C(v->numel()-1,v->numel()-1,outputDerived->rows)]=0.f;
	}
}

__global__ void cuLinear(int numel, float* v, float* output) {
	int vIdx=threadIdx.x;

	while(vIdx<numel) {
		output[vIdx]=v[vIdx];
		vIdx+=blockDim.x;
	}
}

__global__ void cuLinearDerived(int numel, float* v, float* outputDerived,int rows) {
	int vIdx=threadIdx.x;
	
	while(vIdx<numel) {
		outputDerived[IDX2C(vIdx,vIdx,rows)]=1;
		vIdx+=blockDim.x;
	}
}

void linear(vector_ptr v, vector_ptr output, matrix_ptr outputDerived, bool last) {
/*
	for(int i=0;i<v->numel();i++) {
		outputDerived->d_data[IDX2C(i,i,outputDerived->rows)]=1;
		output->d_data[i]=v->d_data[i];
	}
*/
	cuLinear<<<1,16>>>(v->numel()-(last?0:1),thrust::raw_pointer_cast(v->d_data.data()),thrust::raw_pointer_cast(output->d_data.data()));
	cuLinearDerived<<<1,16>>>(v->numel()-(last?0:1),thrust::raw_pointer_cast(v->d_data.data()),thrust::raw_pointer_cast(outputDerived->d_data.data()), outputDerived->rows);
	if(!last) {
		output->d_data[v->numel()-1]=1.f;
		outputDerived->d_data[IDX2C(v->numel()-1,v->numel()-1,outputDerived->rows)]=0.f;
	}
}

typedef void (*neuron_func_t)(vector_ptr, vector_ptr,matrix_ptr,bool last);

neuron_func_t neuron_func[]={sigmoid,linear};


void cublasMul(float alpha,float beta, matrix_ptr m1,hipblasOperation_t transa, matrix_ptr m2, hipblasOperation_t transb, matrix_ptr p) {
	int check1=(transa==HIPBLAS_OP_N?m1->cols:m1->rows);
	int check2=(transb==HIPBLAS_OP_N?m2->rows:m2->cols);

	int lda=m1->rows;
	int ldb=m2->rows;

//	printf("%d %d\n", check1,check2);
	assert(check1==check2);

	p->rows=(transa==HIPBLAS_OP_N?m1->rows:m1->cols);
	p->cols=(transb==HIPBLAS_OP_N?m2->cols:m2->rows);


	hipblasStatus_t status = hipblasSgemm (handle, transa, transb, p->rows, p->cols, check1, &alpha, thrust::raw_pointer_cast (m1->d_data.data()), lda, thrust::raw_pointer_cast (m2->d_data.data()), ldb, &beta, thrust::raw_pointer_cast (p->d_data.data()), p->rows);
	if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "!!!! kernel execution error.\n";
    }
}

inline void cublasMul(matrix_ptr m1, hipblasOperation_t transa, matrix_ptr m2, hipblasOperation_t transb, matrix_ptr p) {
	cublasMul(1.0f,0.f,m1, transa, m2, transb, p);
}
inline void cublasMul(matrix_ptr m1, matrix_ptr m2, matrix_ptr p) {
	cublasMul(1.0f,0.f,m1, HIPBLAS_OP_N, m2, HIPBLAS_OP_N, p);
}
inline void cublasMul(float alpha,float beta,matrix_ptr m1, matrix_ptr m2, matrix_ptr p) {
	cublasMul(alpha,beta,m1, HIPBLAS_OP_N, m2, HIPBLAS_OP_N, p);
}

inline void cublasSub(matrix_ptr m1, matrix_ptr m2, matrix_ptr p) {

	assert(m1->rows==m2->rows);
//	printf("%d %d\n",m1->cols, m2->cols);
	assert(m1->cols==m2->cols);

	int lda=m1->rows;
	int ldb=m2->rows;
	float alpha=1.f;
	float beta=-1.f;
	hipblasStatus_t status=hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m1->rows, m2->cols, &alpha, thrust::raw_pointer_cast(m1->d_data.data()), lda, &beta, thrust::raw_pointer_cast(m2->d_data.data()), ldb, thrust::raw_pointer_cast(p->d_data.data()), p->rows);
}



class Layer {
public:
	int neuronType;
	int neuronNum;
	int bias;
	layer_ptr nextLayer;
	layer_ptr prevLayer;
	matrix_ptr nextMatrix;
	matrix_ptr prevMatrix;
	
	vector_ptr output;
	matrix_ptr outputDerived;
	vector_ptr delta;
	vector_ptr error_derived;
	matrix_ptr unit;
	vector_ptr input;
	matrix_ptr weightAdj;

	Layer(int _neuronNum, int _neuronType, bool _bias=true) : neuronNum(_neuronNum), neuronType(_neuronType), bias(_bias), nextLayer(0), prevLayer(0), weightAdj(0) {
		cout << __PRETTY_FUNCTION__ << _neuronNum << endl;
/*
		if(bias) {
			output=new RowVector(neuronNum+1);
			output->d_data[neuronNum]=1.f;
			outputDerived=new Matrix(neuronNum,neuronNum+1);
		} else {
*/
			output=new RowVector(neuronNum);
			outputDerived=new Matrix(neuronNum,neuronNum);
/*
		}
*/
		input=new RowVector(neuronNum);
		delta=new ColVector(neuronNum);
		error_derived=new RowVector(neuronNum);
		unit=new Matrix(neuronNum);

	}	

	virtual ~Layer() {
		delete  output;
		cout << __PRETTY_FUNCTION__ << neuronNum << endl;
	}

	void excite() {
		if(prevLayer!=0) {
			cublasMul(prevLayer->output, prevMatrix, input);
			neuron_func[neuronType](input,output, outputDerived,nextLayer==0);
		}
		if(nextLayer!=0)
			nextLayer->excite();
	}
	layer_ptr error(vector_ptr desiredOutput) {
		if(prevLayer!=0) {
			cublasSub(output,desiredOutput,error_derived);
			cublasMul(outputDerived, HIPBLAS_OP_N, error_derived, HIPBLAS_OP_T, delta);
			
		}
		return this;
	}

	void backpropagation() {
		if(nextLayer!=0 && prevLayer!=0) {
			Matrix m(outputDerived->rows, nextMatrix->cols);
			cublasMul(outputDerived, HIPBLAS_OP_N, nextMatrix, HIPBLAS_OP_N, &m);
			cublasMul(&m,nextLayer->delta, delta);
		}
		if(prevLayer!=0)
			prevLayer->backpropagation();
	}

	void adjust() {
		if(prevLayer!=0) {
			if(weightAdj==0)
				weightAdj=new Matrix(prevMatrix->cols, prevMatrix->rows);
			cublasMul(LEARNING_RATE, 0.0f, delta, HIPBLAS_OP_N, prevLayer->output, HIPBLAS_OP_N, weightAdj);
			cublasMul(-1.f, 1.f, weightAdj, HIPBLAS_OP_T, unit, HIPBLAS_OP_N, prevMatrix);
		}
		if(nextLayer!=0)
			nextLayer->adjust();
	}

	void adjustAdd() {
		if(prevLayer!=0) {
			cublasMul(-1.f, 1.f, weightAdj, HIPBLAS_OP_T, unit, HIPBLAS_OP_N, prevMatrix);
			cublasMul(0.f,0.f, delta, HIPBLAS_OP_N, prevLayer->output, HIPBLAS_OP_N, weightAdj);
		}
		if(nextLayer!=0)
			nextLayer->adjustAdd();
	}

	void addTail(layer_ptr layer) {
		layer_ptr last=this;
		while(last->nextLayer!=0) {
			last=last->nextLayer;
		}
		last->nextLayer=layer;
		layer->prevLayer=last;
		
	}
};

struct Autoencoder {
	layer_ptr inputLayer;
	layer_ptr hiddenLayer;
	layer_ptr outputLayer;

	autoencoder_ptr next;
	autoencoder_ptr prev;
	
	Autoencoder(int _inputNum, int _hiddenNum) : next(0), prev(0) {
		inputLayer=new Layer(_inputNum, SIGMOID_0);
		hiddenLayer=new Layer(_hiddenNum, SIGMOID_0);
		outputLayer=new Layer(_inputNum, LINEAR,false);

		inputLayer->addTail(hiddenLayer);
		inputLayer->addTail(outputLayer);

		int bias=1;
		for(layer_ptr i=inputLayer;i!=outputLayer;i=i->nextLayer) {
			i->nextMatrix=new Matrix(i->neuronNum, i->nextLayer->neuronNum);
			i->nextLayer->prevMatrix=i->nextMatrix;
			i->nextMatrix->randomize();
		}	
	}

	
	void addTail(autoencoder_ptr autoencoder) {
		autoencoder_ptr last=this;
		while(last->next!=0) {
			last=last->next;
		}
		last->next=autoencoder;
		autoencoder->prev=last;
	}

	void excite(vector_ptr p) {
		inputLayer->output->set(*p);
		inputLayer->excite();
	}
	
	void train(vector_ptr p) {
		for(int i=0;i<1;i++) {
				excite(p);
				outputLayer->error(p)->backpropagation();
				inputLayer->adjust();
		}
//		inputLayer->adjustAdd();
/*
		if(next!=0) {
			next->train(hiddenLayer->output);
		}
*/
	}
};


struct NeuralNet {
	layer_ptr inputLayer;
	layer_ptr outputLayer;
	vector_ptr error;
	autoencoder_ptr autoencoder;
	autoencoder_ptr autoencoder_pretrain;
	int bias;

	NeuralNet() {};
public:
	NeuralNet(int _inputNum,int _outputNum, int _hiddenLayerNum, std::vector<int> layerNeuronNum) : autoencoder(0), bias(1){
		cout << __PRETTY_FUNCTION__ << endl;
		error=new ColVector(_outputNum);
		printf("_inputNum %d\n", _inputNum);
		
		inputLayer=new Layer(_inputNum, SIGMOID_0);
		for(int i=0; i<_hiddenLayerNum;i++) {

			if(autoencoder==0) {
				autoencoder=new Autoencoder(_inputNum, layerNeuronNum[i]);
			} else {
				autoencoder->addTail(new Autoencoder(layerNeuronNum[i-1], layerNeuronNum[i]));
			}

			inputLayer->addTail(new Layer(layerNeuronNum[i], SIGMOID_0));
		}

		outputLayer=new Layer(_outputNum, LINEAR,false);
		inputLayer->addTail(outputLayer);
		autoencoder->addTail(new Autoencoder(layerNeuronNum[_hiddenLayerNum-1],_outputNum));

		autoencoder_ptr a=autoencoder;

		for(layer_ptr i=inputLayer;i!=outputLayer;i=i->nextLayer) {
//			i->nextMatrix=new Matrix(i->neuronNum, i->nextLayer->neuronNum);
			i->nextMatrix=a->inputLayer->nextMatrix;
			i->nextLayer->prevMatrix=i->nextMatrix;
			i->nextMatrix->randomize();
			if(a!=0)
				a=a->next;
		}
		autoencoder_pretrain=autoencoder;
	}

	void pretrain(vector_ptr p) {
		autoencoder_ptr a=autoencoder;
		if(a==autoencoder_pretrain) {
			autoencoder_pretrain->train(p);
		} else {
				a->excite(p);
				a=a->next;
				while(a!=autoencoder_pretrain) {
					a->excite(a->prev->hiddenLayer->output);
					a=a->next;
				}
				autoencoder_pretrain->train(autoencoder_pretrain->prev->hiddenLayer->output);
		}
	}

	void pretrainNext() {
		autoencoder_pretrain=autoencoder_pretrain->next;
	}

	void pretrainAdjust() {
		autoencoder_pretrain->inputLayer->adjustAdd();
	}

	
	vector_ptr excite(vector_ptr p) {
		inputLayer->output->set(*p);
		inputLayer->excite();
/*
		layer_ptr layer=inputLayer->nextLayer;
		layer->excite();
*/
/*
		while(layer!=0) {
			layer->excite();
			layer=layer->nextLayer;
		}
*/
		return outputLayer->output;
	}

	void backpropagation(vector_ptr desiredOutput) {
		layer_ptr layer=outputLayer;
		layer->error(desiredOutput)->backpropagation();
/*
		layer=layer->prevLayer;
		layer->backpropagation();
*/

/*
		while(layer!=0) {
			layer->backpropagation();
			layer=layer->prevLayer;
		}
*/
		inputLayer->adjust();
	}

	void adjust() {
		inputLayer->adjustAdd();
	}

	virtual ~NeuralNet() {
/*
		for(layer_ptr i=outputLayer; i!=0UL;i=i->prevLayer) {
			delete i->nextMatrix;
			delete i->nextLayer;
		}
		delete inputLayer;
		delete error;
*/
		cout << __PRETTY_FUNCTION__ << endl;
	}
} nn_t;


// ----------------------------------------------------------------------------------------------------
//                       main function
// ----------------------------------------------------------------------------------------------------

void usage(char** argv) {
	fprintf(stderr,"Usage %s [-g genome] [-i indata] [-o outdata] [-s samples] [-n input_size] [-u output_size]\n", argv[0]);
	exit(-1);

}

int main(int argc, char **argv) {
	srand(time(0));
	int opt;
	if(argc!=13) {
		usage(argv);
	}
	const char *filename, *infile, *outfile;
	int samples, input_size, output_size;

	while((opt=getopt(argc, argv, "i:o:g:s:n:u:"))!=-1) {
		switch(opt) {
			case 'g':
				filename=optarg;
				break;
			case 'i':
				infile=optarg;
				break;
			case 'o':
				outfile=optarg;
				break;
			case 's':
				samples=atoi(optarg);
				break;
			case 'n':
				input_size=atoi(optarg);
				break;
			case 'u':
				output_size=atoi(optarg);
				break;
			default:
				usage(argv);
		}
	}
	IO io(input_size, output_size, samples,infile, outfile);

	hipblasStatus_t status=hipblasCreate(&handle);
	if(status!=HIPBLAS_STATUS_SUCCESS) {
		cerr << "cublas init failed" << endl;
	}
	Matrix m1(3,2);
	m1.d_data[IDX2C(0,0,3)]=1;
	m1.d_data[IDX2C(1,0,3)]=2;
	m1.d_data[IDX2C(2,0,3)]=3;

	m1.d_data[IDX2C(0,1,3)]=2;
	m1.d_data[IDX2C(1,1,3)]=3;
	m1.d_data[IDX2C(2,1,3)]=4;

	Matrix m2(2,3);
	m2.d_data[IDX2C(0,0,2)]=2;
	m2.d_data[IDX2C(1,0,2)]=3;

	m2.d_data[IDX2C(0,1,2)]=3;
	m2.d_data[IDX2C(1,1,2)]=4;

	m2.d_data[IDX2C(0,2,2)]=4;
	m2.d_data[IDX2C(1,2,2)]=5;

	Matrix p(3,2);
	Matrix m3(2);

	cublasMul(&m1,&m3, &p);
	p.print();
//	exit(0);

/*
	for(int i=0;i<p.rows;i++) {
		for(int j=0;j<p.cols;j++)
			cout << p.d_data[IDX2C(i,j,p.rows)] << ",";
		cout << endl;
	}
*/

	int d[]={15+1,8+1,8+1,8+1,8+1,8+1,8+1,8+1};
	int l=sizeof(d)/sizeof(int);
	NeuralNet nn(input_size,output_size,l,std::vector<int>(d, d+l));

	RowVector v(input_size);
	RowVector v1(output_size);
#if 1
	cout << "pretraining ..." << endl;
	for(int r=0;r<l;r++) {
			cout << "autoencoder " << r << endl;
			for(int k=0;k<300;k++) {
					for(int i=0;i<samples;i++) {
						for(int j=0;j<input_size;j++) {
							v.set(j,io.input[IDX2C(j,i,input_size)]);
						}
						for(int j=0;j<output_size; j++) {
							v1.set(j, io.output[IDX2C(j,i,output_size)]);
						}
						nn.pretrain(&v);
					}
//					nn.pretrainAdjust();
			}
			nn.pretrainNext();
	}
#endif
	cout << "training ..." << endl;
for(int k=0;k<40000;k++) {
if(k%50==0)
		cout << k << endl;
	for(int i=0;i<samples;i++) {
		for(int j=0;j<input_size;j++) {
			v.set(j,io.input[IDX2C(j,i,input_size)]);
		}
		for(int j=0;j<output_size; j++) {
			v1.set(j, io.output[IDX2C(j,i,output_size)]);
		}
		vector_ptr o=nn.excite(&v);
		if(k%50==0) {
				cout << "input: ";
				v.printd();
				cout << " output: ";
				v1.printd();
		//		cout << endl;
				cout << "fit: ";
				o->print();
				cout << " ";
				int ham=0;
				for(int r=0;r<output_size;r++) {
					int a=o->d_data[r]>.5?1:0;
					int b=v1.d_data[r]>.5?1:0;
					ham+=a^b;
					printf("%d,", a);
				}
				cout << " distance: " << ham;
				cout << endl;
		}

		nn.backpropagation(&v1);
	}
//	nn.adjust();
	if(k%50==0)
			cout << "-------------------------------------------------------------------------" << endl;
}

	status=hipblasDestroy(handle);
	if(status!=HIPBLAS_STATUS_SUCCESS) {
		cerr << "cublas shutdown failed" << endl;
	}
	return 0;
}
