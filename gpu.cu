#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <iostream>
#include <queue>		// std::priority_queue
#include <math.h>
#include <stdio.h>
#include <sys/time.h>

#include <hiprand/hiprand_kernel.h>
#include <assert.h>

using namespace std;

#define DEBUG

#define NEURON_NUM 16
#define CORTEX_NUM 8
#define POPULATION_SIZE 256
#define MUTATION_PROB 0.15
#define CROSSOVER_PROB 0.8
#define BEST_INDIVIDUALS 2

#define NR_INPUTS 5
#define NR_OUTPUTS 15
#define SAMPLES 16

#define CHAR(x) ((char*)x)
#define TO_WEIGHT(x) (((x&0x0f)-8.)/7.)
#define WEIGHT(w,x,y) TO_WEIGHT((w[(y*NEURON_NUM+x)>>1]>>(((y*NEURON_NUM+x)<<2)&0x7))&0xf)
#define INTERCONNECT_INDEX(i,c,n) (((c*NEURON_NUM+n)*16/8)>>2)
#define INTERCONNECT_VALUE(i,c,n) i[c][n]/*((((((unsigned int*)(i))[INTERCONNECT_INDEX(i,c,n)]))>>(((c*NEURON_NUM+n)*16)%32)))*/
#define INTERCONNECT_WEIGHT(i,c,n) TO_WEIGHT((INTERCONNECT_VALUE(i,c,n)>>8)&0xf)
#define INTERCONNECT_CORTEX(i,c,n) (INTERCONNECT_VALUE(i,c,n)&0xf)
#define INTERCONNECT_NEURON(i,c,n) ((INTERCONNECT_VALUE(i,c,n)>>4)&0xf)


#define FITNESS(population, gIdx) ((population)->fitness[gIdx])
#define THREAD_ID (blockIdx.x*blockDim.x + threadIdx.x)
#define _THREAD_COUNT (blockDim.x*gridDim.x)

#define NEURON_IDX(output_neuron_idx) (output_neuron_idx&0xf)
#define CORTEX_IDX(output_neuron_idx) ((output_neuron_idx>>4)&0xf)


typedef struct __attribute__ ((__packed__)) {
	char weight[NEURON_NUM*NEURON_NUM/2]; // weight is 4 bits
} cortex_t;

typedef struct __attribute__ ((__packed__)) {
	cortex_t cortex[CORTEX_NUM];
	short interconnect[CORTEX_NUM][NEURON_NUM]; // 16 bit value, use only 12
	unsigned char input_weight[NR_INPUTS][CORTEX_NUM][NEURON_NUM];
	unsigned char output_neuron_idx[NR_OUTPUTS];
} genome_t;


typedef struct {
	genome_t genome[POPULATION_SIZE];
	float fitness[POPULATION_SIZE];
	float neuron_output[POPULATION_SIZE][CORTEX_NUM][NEURON_NUM];
	float input[NR_INPUTS];
	float output[NR_OUTPUTS];
	float error[POPULATION_SIZE];
	float outputs[POPULATION_SIZE][NR_OUTPUTS];
	
	hiprandState_t hiprandState;
	int map[POPULATION_SIZE];
} population_t;

typedef struct {
    float inputs[NR_INPUTS][SAMPLES];
    float outputs[NR_OUTPUTS][SAMPLES];
    float errors[NR_OUTPUTS];
} IO_t;

struct Counter {
	int count;
	__device__ Counter() : count(0) {};
    __device__ int getPopulationIndex() const { int r=_THREAD_COUNT*count+THREAD_ID; return r;};
    __device__ int getPopulationIndexInc() { int r=getPopulationIndex(); count++; return r; };
    __device__ int getPopulationIndexEven() { int r=_THREAD_COUNT*count+THREAD_ID; return r; };
    __device__ int getPopulationIndexOdd() { int r=_THREAD_COUNT*(count+1)+THREAD_ID; return r; };
    __device__ void reset() { count=0; };
};

float *deviceBestIndividualFitness;
float *hostBestIndividualFitness;

//---------------------------------------------------------------------------------------------------

inline void check_cuda_errors(const char *filename, const int line_number)
{
      hipDeviceSynchronize();
      hipError_t error = hipGetLastError();
      if(error != hipSuccess) {
          printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
          exit(-1);
      }
}

__device__ inline float sigmoid(float signal) {
	return signal / sqrtf(1+powf(signal, 2));
}

__device__ 
void
swap (int array[], int i, int j)
{
  int tmp = array[i];
  array[i] = array[j];
  array[j] = tmp;
}

__device__ void
qs (population_t *population, int array[], int left, int right)
{
  if (left < right)
    {
      double p = FITNESS(population, array[left + (right - left) / 2]);
      int i = left;
      int j = right;

      while (i < j)
	{

	  while (FITNESS(population,array[i]) > p && i < right)
	    i++;
	  while (FITNESS(population,array[j]) < p && j > left)
	    j--;
	  if (i <= j)
	    {
	      swap (array, i, j);
	      i++;
	      j--;
	    }
	}

      qs (population,array, i, right);
      qs (population,array, left, j);
    }

}

__device__ void qsort(population_t *population) {
    qs(population, population->map, 0, POPULATION_SIZE-1);
}

__global__ void cuReset(population_t *p) {
	if(THREAD_ID==0) {
		memset(p->fitness, 0, sizeof(p->fitness));
		memset(p->neuron_output, 0, sizeof(p->neuron_output));
		memset(p->input, 0, sizeof(p->input));
		memset(p->output, 0, sizeof(p->output));
		memset(p->error, 0, sizeof(p->error));
		memset(p->outputs, 0, sizeof(p->outputs));
		for(int i=0;i<POPULATION_SIZE;i++) {
			p->map[i]=i;
		}
	}
		
}

__global__ void cuResetError(population_t *p) {
	int g=blockIdx.x;
	while(g < POPULATION_SIZE) {
		p->error[g]=0.f;
		g+=gridDim.x;
	}
}

__global__ void cuResetNeurons(population_t *p) {
	int g=blockIdx.x;

	while(g < POPULATION_SIZE) {
		memset(p->neuron_output[g], 0, sizeof(float)*CORTEX_NUM*NEURON_NUM);
		g+=gridDim.x;
	}
}


__global__ void cuRandInit(population_t *p) {
    	hiprand_init(2345,THREAD_ID, 0, &p->hiprandState);
}

void hipInit(population_t* p1, population_t *p2) {
		cuRandInit<<<1,1>>>(p1);
		cuReset<<<1,1>>>(p1);
		cuRandInit<<<1,1>>>(p2);
		cuReset<<<1,1>>>(p2);
}

__device__ inline float my_random (population_t *p) {
  return ((float) (hiprand (&p->hiprandState) / ((float)(0x0FFFFFFFFUL))));
}

__device__ void mutate (population_t *p, double prob, genome_t * genome) {
  double r = my_random (p);
  int s = hiprand(&p->hiprandState) % (sizeof (genome_t) * 8);
  if (r < prob)
    {
      ((char *) genome)[s / 8] ^= (1 << (s % 8));
    }
}

__device__ void cross(char* ng1, char* g1,char* ng2, char* g2, int s, int size) {

      memcpy (ng1, g1, s / 8);
      memcpy (CHAR (ng1) + s / 8, CHAR (g2) + s / 8, size - s / 8);
      char mask = (1 << (s % 8)) - 1;
      CHAR (ng1)[s / 8] = (CHAR (g1)[s / 8] & ~mask) | (CHAR (g2)[s / 8] & mask);

      memcpy (ng2, g2, s / 8);
      memcpy (CHAR (ng2) + s / 8 , CHAR (g1) + s / 8 , size - s / 8);
      mask = (1 << (s % 8)) - 1;
      CHAR (ng1)[s / 8] = (CHAR (g2)[s / 8] & ~mask) | (CHAR (g1)[s / 8] & mask); 
}

__device__ void _crossover (Counter &c, double prob, population_t * population, population_t * new_population) {


  double r = my_random (population);
  double total = 0;
  assert(c.getPopulationIndex() < POPULATION_SIZE);
  genome_t *ng1 = &new_population->genome[c.getPopulationIndexInc()];
  assert(c.getPopulationIndex() < POPULATION_SIZE);
  genome_t *ng2 = &new_population->genome[c.getPopulationIndexInc()];


  for (int i = 0; i < POPULATION_SIZE; i++)
    {
      total += population->fitness[i];
    }
  genome_t *g1 = 0;
  genome_t *g2 = 0;
  while (g1 == g2) {
      double r1 = my_random (population) * total;
      double r2 = my_random (population) * total;
      double sum = 0;
      for (int i = 0; i < POPULATION_SIZE; i++)
	{
	  sum += population->fitness[i];
	  if (sum >= r1)
	    {
	      g1 = &population->genome[i];
	      break;
	    }
	}
      sum = 0;
      for (int i = 0; i < POPULATION_SIZE; i++)
	{
	  sum += population->fitness[i];
	  if (sum >= r2)
	    {
	      g2 = &population->genome[i];
	      break;
	    }
	}
  }

  if (r < prob)
    {

       int s= hiprand(&population->hiprandState) % (sizeof(genome_t)*8);

       //genome_t ngg1,ngg2;

       cross((char*)ng1, (char*)g1,(char*)ng2,(char*)g2,s,sizeof(genome_t));
       /*
       cross((char*)&ngg1, (char*)&g1->genome,(char*)&ngg2,(char*)&g2->genome,s,sizeof(genome_t));

       int s1 = (s==0?0:(hiprand(&THREAD(thread).hiprandState) % s));
       int new_size=(s+0)/8;

       if(new_size>0)
           cross((char*)&ng1->genome, (char*)&ngg1,(char*)&ng2->genome,(char*)&ngg2,s1,new_size);

       int s2 = hiprand(&THREAD(thread).hiprandState) % (sizeof(genome_t)*8-s);

       if(sizeof(genome_t)-new_size>0)
           cross(((char*)&ng1->genome)+new_size, ((char*)&ngg1)+new_size,((char*)&ng2->genome)+new_size,((char*)&ngg2)+new_size,s2,sizeof(genome_t)-new_size);
           */
    }
  else
    {
      memcpy (ng1, g1, sizeof (genome_t));
      memcpy (ng2, g2, sizeof (genome_t));
    }

  mutate (population, MUTATION_PROB, ng1);
  mutate (population, MUTATION_PROB, ng2);
}

__global__ void crossover (double prob, population_t * population, population_t * new_population) {


	Counter c;
    if(THREAD_ID==0) {
		for(int i=0; i<BEST_INDIVIDUALS && i<POPULATION_SIZE;i++) {
	        c.getPopulationIndexInc();
		}
    }

    while(c.getPopulationIndexEven() < POPULATION_SIZE && c.getPopulationIndexOdd() < POPULATION_SIZE) {
//		if(c.getPopulationIndexEven()==0)
//		printf("cross na %d %d\n", c.getPopulationIndexEven(), c.getPopulationIndexOdd());
	     _crossover(c, prob, population, new_population);
    }

}

__global__ void cuInputs(population_t *p, IO_t *io,int sample) {
	p->input[threadIdx.x]=io->inputs[threadIdx.x][sample];
//	printf("%d %f,", threadIdx.x, p->input[threadIdx.x]);
}

__global__ void cuOutputs(population_t *p, IO_t *io, int sample) {
	p->output[threadIdx.x]=io->outputs[threadIdx.x][sample];
//	printf("%d %f,", threadIdx.x, p->output[threadIdx.x]);
}

__global__ void cuExcite(population_t* p) {
	int g=blockIdx.x;

	int cIdx=threadIdx.x;
	int nIdx=threadIdx.y;


//	printf("cIdx: %d nIdx: %d\n", cIdx,nIdx);

	while(g < POPULATION_SIZE) {
//		__shared__ float neuron_output[CORTEX_NUM][NEURON_NUM];
		genome_t *genome=genome=&p->genome[g];

//		neuron_output[cIdx][nIdx]=p->neuron_output[g][cIdx][nIdx];
		__syncthreads();

		float signal=0.;
		// toto je vypocet jedneho cortexu
		for(int n=0;n<NEURON_NUM;n++) {
			float factor=WEIGHT(genome->cortex[cIdx].weight, nIdx, n);
			signal+=factor*p->neuron_output[g][cIdx][n];
		}
		__syncthreads();
		// toto su signali z ostatnych cortexov
		for(int n=0;n<CORTEX_NUM;n++) {
			for(int m=0;m<NEURON_NUM;m++) {
				if(INTERCONNECT_CORTEX(genome->interconnect,n,m)==cIdx)
					if( INTERCONNECT_NEURON(genome->interconnect,n,m)==nIdx) {
					float factor=INTERCONNECT_WEIGHT(genome->interconnect,n,m);
					signal+=factor*p->neuron_output[g][n][m];
//					printf("%f\n", p->neuron_output[g][n][m]);
				}
			}
		}
		__syncthreads();
		// toto su signali z zo vstupov
		for(int n=0;n<NR_INPUTS;n++) {
			float factor=TO_WEIGHT(genome->input_weight[n][cIdx][nIdx]);
//			printf("p input: %f %f %d\n", p->input[n],factor, n);
			signal+=factor*p->input[n];
		}

/*
		__syncthreads();
		neuron_output[cIdx][nIdx]=sigmoid(signal);
*/

		__syncthreads();	
		p->neuron_output[g][cIdx][nIdx]=sigmoid(signal);//neuron_output[cIdx][nIdx];
		g+=gridDim.x;
	}
}

__global__ void cuError(population_t *p) {
	int g=blockIdx.x;
	int oIdx=threadIdx.x%NR_OUTPUTS;

	while(g < POPULATION_SIZE) {
		__shared__ float error[NR_OUTPUTS];
		genome_t *genome=&p->genome[g];
		int nIdx=NEURON_IDX(genome->output_neuron_idx[oIdx]);
		int cIdx=CORTEX_IDX(genome->output_neuron_idx[oIdx]);
		float output=p->neuron_output[g][cIdx][nIdx];
		p->outputs[g][oIdx]=output;
        int b1=p->output[oIdx]>0.?1:0;
        int b2=output>0.?1:0;
		error[oIdx]=.9*(b1^b2)+.1*powf(p->output[oIdx] - output,2);//(SAMPLES*NR_OUTPUTS);
		
		__syncthreads();
/*
		for(int stride=blockDim.x>>1;stride>0;stride>>=1) {
			if(threadIdx.x<stride) {
				error[threadIdx.x]+=error[threadIdx.x+stride];
			}
		}
*/
		__syncthreads();
		if(threadIdx.x==0) {
			float e=0.;
			for(int i=0;i<NR_OUTPUTS;i++)
				e+=error[i];
			p->error[g]+=e;
		}
		__syncthreads();

		g+=gridDim.x;
	}
}

__global__ void cuFitness(population_t *p) {
	int g=blockIdx.x;

	while(g < POPULATION_SIZE) {
		p->fitness[g]=1./(p->error[g]+0.00001);
		g+=gridDim.x;
	}
}

__global__ void init_random_population (population_t* currentPopulation)
{
  int g=blockIdx.x;
  while(g < POPULATION_SIZE) {
      for (int j = 0; j < sizeof (genome_t); j++) {
    	  ((char *) &(currentPopulation->genome[g]))[j] = hiprand (&currentPopulation->hiprandState) % ~(0U);
	}
	g+=gridDim.x;
 }
}

__global__ void find_best_individual(population_t* population, float *deviceBestIndividualFitness) {
  if(THREAD_ID==0) {
    qsort(population);
    *deviceBestIndividualFitness=population->fitness[population->map[0]];
//	for(int i=0;i<20;i++) 
//		printf("%f,",population->fitness[population->map[i]]);
  }
}

__global__ void copy_best_individuals (population_t * p1, population_t * p2)
{
    if(THREAD_ID==0) {
		Counter c;
        for(int i=0;i<BEST_INDIVIDUALS && i<POPULATION_SIZE;i++) {
		  int pi=c.getPopulationIndexInc();
          memcpy (&p2->genome[pi], &p1->genome[p1->map[i]], sizeof (genome_t));
//		printf("copy na %d\n", pi);
		  
		  
        }
    }
}



void fitness(population_t *population1,IO_t *io, int sample) {
		cuInputs<<<1,NR_INPUTS>>>(population1, io,sample);
		check_cuda_errors(__FILE__, __LINE__);
		cuOutputs<<<1,NR_OUTPUTS>>>(population1, io,sample);
		check_cuda_errors(__FILE__, __LINE__);
		cuExcite<<<32,dim3(CORTEX_NUM,NEURON_NUM)>>>(population1);
		check_cuda_errors(__FILE__, __LINE__);
		cuError<<<128,NR_OUTPUTS>>>(population1);
		check_cuda_errors(__FILE__, __LINE__);
		cuFitness<<<128,1>>>(population1);
		check_cuda_errors(__FILE__, __LINE__);
}

__global__ void print_outputs(population_t *p, IO_t *io,int sample) {
		int b=p->map[0];
		printf("input: ");
		for(int j=0;j<NR_INPUTS;j++) {
			printf("%d,", p->input[j]>0?1:0);
		}
		int ham=0;
		printf("\noutput: [");
		for(int j=0;j<NR_OUTPUTS;j++) {
			int out=p->outputs[b][j]>0?1:0;
			ham+=out^(io->outputs[j][sample]>0?1:0);
			printf("%d,", out);
		}
		printf("]  ");
		for(int j=0;j<NR_OUTPUTS;j++) {
			printf("%f,", p->outputs[b][j]);
		}
		printf(" err: %f ", p->error[b]);
		printf("fitness: %f", p->fitness[b]);
		printf(" distance: %d\n", ham);
}

void print_best(population_t *population, IO_t *io, float *deviceBestIndividualFitness, int start_sample, int stop_sample) {
	cuResetNeurons<<<128,1>>>(population);
	cuResetError<<<128,1>>>(population);
	for(int i=start_sample;i<stop_sample;i++) {
		fitness(population, io, i);
		find_best_individual<<<1,1>>>(population, deviceBestIndividualFitness);
		print_outputs<<<1,1>>>(population,io, i);
	}
}

//---------------------------------------------------------------------------------------------

void genetic (IO_t *io,population_t* population1, population_t* population2, float *deviceBestIndividualFitness) {


      int start_sample=0;
      int stop_sample=1;
      int stride=1;

	  hipInit(population1, population2);

  	  init_random_population<<<128,1>>> (population1);
  	  init_random_population<<<128,1>>> (population2);

      while(stride <= SAMPLES) {
          int g=0;
          stop_sample=stride;
          start_sample=stride-stop_sample;

        while(stop_sample<=SAMPLES) {  
              printf("training %d-%d ...\n", start_sample, stop_sample-1);

          int it=0;

          do {
			  find_best_individual<<<1, 1>>>(population1,deviceBestIndividualFitness);
              copy_best_individuals<<<128,1>>> (population1, population2);
              check_cuda_errors(__FILE__, __LINE__);
              cuResetNeurons<<<128,1>>> (population2); 
              cuResetError<<<128,1>>> (population2); 
              check_cuda_errors(__FILE__, __LINE__);

			  if(it>0) // kvoli pocitaniu fitness, pri 0 este nie je vypocitany
				crossover<<<128,1>>> (CROSSOVER_PROB, population1, population2);
              check_cuda_errors(__FILE__, __LINE__);


              {
                  population_t *p = population1;
                  population1 = population2;
                  population2 = p;
              }

			  for(int i=start_sample; i<stop_sample;i++) {
				  fitness(population1, io, i);
			  }

              if(g%10==0) {
			  find_best_individual<<<1, 1>>>(population1,deviceBestIndividualFitness);
			  	  printf("generation %d\n", it);
                  check_cuda_errors(__FILE__, __LINE__);
                  hipMemcpy(hostBestIndividualFitness, deviceBestIndividualFitness, sizeof(double), hipMemcpyDeviceToHost);
				  print_best(population1, io, deviceBestIndividualFitness, start_sample, stop_sample);
                  check_cuda_errors(__FILE__, __LINE__);
				  printf("best fitness: %f\n", *hostBestIndividualFitness);
              }

            g++;
            it++;
          } while (*hostBestIndividualFitness < 2);
	  		*hostBestIndividualFitness=0.;

          stop_sample+=stride;
          start_sample=stop_sample-stride;
        }
        stride<<=1;
    }
    start_sample=0;

}


void read_io(IO_t &io) {
#if 0
  const char* in[]={
#include "switch_in.dat"
      };
  const char* out[]={
#include "switch_out.dat"
      };
#else
      const char* in[]={
#include "bch_input.dat.2"
      };
      const char* out[]={
#include "bch_output.dat.2"
      };
#endif
  for(int k=0;k<SAMPLES;k++) {
      sscanf(in[k],"%f,%f,%f,%f,%f", 
        &io.inputs[0][k],
        &io.inputs[1][k],
        &io.inputs[2][k],
        &io.inputs[3][k],
        &io.inputs[4][k]
        );
  //    cout << io.inputs[0][k] <<"," << io.inputs[1][k] << "," << io.inputs[2][k] << endl;
  }
  for(int k=0;k<SAMPLES;k++) {
      sscanf(out[k],"%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f", 
        &io.outputs[0][k],
        &io.outputs[1][k],
        &io.outputs[2][k],
        &io.outputs[3][k],
        &io.outputs[4][k],
        &io.outputs[5][k],
        &io.outputs[6][k],
        &io.outputs[7][k],
        &io.outputs[8][k],
        &io.outputs[9][k],
        &io.outputs[10][k],
        &io.outputs[11][k],
        &io.outputs[12][k],
        &io.outputs[13][k],
        &io.outputs[14][k]
        );
   //   cout << io.outputs[0][k] <<"," << io.outputs[1][k] << endl;
  }
      /*
  memset(io.inputs,0, sizeof(io.inputs));
  for(int k=0;k<=SAMPLES;k++) {
      for(int l=0;l<20;l++) {
          io.inputs[keno_input2[k][l]-1][k]=1;
      }
  }
  memset(io.outputs,0, sizeof(io.outputs));
  for(int k=0;k<SAMPLES;k++) {
      for(int l=0;l<20;l++) {
          io.outputs[keno_output2[k][l]-1][k]=1;
      }
  }
  cout << "sizeof " << sizeof(io.inputs) << "/" << sizeof(double)  << endl;
  */

}


int main() {


	population_t *p1,*p2;
	IO_t tmp;
	IO_t *io;

	read_io(tmp);
	
	hipMalloc((void**)&p1, sizeof(population_t));
	hipMalloc((void**)&p2, sizeof(population_t));
  	hipMalloc((void**)&deviceBestIndividualFitness, sizeof(double));
	hipHostAlloc((void**)&hostBestIndividualFitness,sizeof(double), hipHostMallocWriteCombined);
	hipMalloc((void**)&io,sizeof(IO_t));

	hipMemcpy(io,&tmp,sizeof(IO_t),hipMemcpyHostToDevice);

	genetic(io, p1,p2, deviceBestIndividualFitness);

	hipFree(p1);
	hipFree(p2);
	hipFree(io);
	hipFree(hostBestIndividualFitness);

	return 0;
}
