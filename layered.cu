#include "layered.h"

const char *filename;

// ----------------------------------------------------------------------------------------------------
//                       main function
// ----------------------------------------------------------------------------------------------------

void usage(char** argv) {
	fprintf(stderr,"Usage %s [-n filename]\n", argv[0]);
	exit(-1);

}
int main(int argc, char **argv) {

	int opt;
	if(argc<2) {
		usage(argv);
	}

	while((opt=getopt(argc, argv, "n:"))!=-1) {
		switch(opt) {
			case 'n':
				filename=optarg;
				break;
			default:
				usage(argv);
		}
	}

	assert(NR_INPUTS<=NEURON_NUM);
	printf("population: %ld\n", sizeof(population_t)/1024/1024);
	printf("ga_genome_t %ld\n", sizeof(ga_genome_t));
	printf("bp_genome_t %ld\n", sizeof(bp_genome_t));

	population_t *p1,*p2;
	IO_t tmp;
	IO_t *io;

	read_io(tmp);
	
	cuAllocPopulation(&p1);
	cuAllocPopulation(&p2);
/*
	hipMalloc((void**)&p1, sizeof(population_t));
	hipMalloc((void**)&p2, sizeof(population_t));
*/
  	hipMalloc((void**)&deviceBestIndividualFitness, sizeof(double));
	hipHostAlloc((void**)&hostBestIndividualFitness,sizeof(double), hipHostMallocWriteCombined);
	hipMalloc((void**)&io,sizeof(IO_t));

	hipMemcpy(io,&tmp,sizeof(IO_t),hipMemcpyHostToDevice);

	hoReset(p1);
	hoReset(p2);
	population_t *p=genetic(io, p1,p2, deviceBestIndividualFitness);

	int fd=open(filename, O_CREAT|O_WRONLY|O_TRUNC, 0444);
	writeBest(fd, p);
	close(fd);

	hipFree(p1);
	hipFree(p2);
	hipFree(io);
	hipFree(hostBestIndividualFitness);
	return 0;
}
